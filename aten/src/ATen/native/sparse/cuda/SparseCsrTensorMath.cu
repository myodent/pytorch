#include <ATen/ATen.h>
#include <ATen/ExpandUtils.h>
#include <ATen/InitialTensorOptions.h>
#include <ATen/NativeFunctions.h>
#include <ATen/SparseCsrTensorImpl.h>
#include <ATen/SparseCsrTensorUtils.h>
#include <ATen/SparseTensorUtils.h>
#include <ATen/WrapDimUtilsMulti.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/Resize.h>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <type_traits>

#include <THC/THCTensorMathPointwise.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAUtils.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <ATen/native/sparse/cuda/SparseCUDABlas.cuh>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>

namespace at {
namespace native {

using namespace at::sparse_csr;
// certain utiliy functions are usable from sparse COO.
using namespace at::sparse;

Tensor& add_out_dense_sparse_csr_cuda(
    Tensor& output,
    const Tensor& dense,
    const SparseCsrTensor& src,
    const Scalar& alpha) {
  TORCH_INTERNAL_ASSERT(dense.layout() == kStrided);
  TORCH_INTERNAL_ASSERT(src.is_sparse_csr());
  TORCH_INTERNAL_ASSERT(dense.is_cuda());

  TORCH_CHECK(
      output.is_contiguous(),
      "out argument must be contiguous, but got: ",
      output.suggest_memory_format());
  TORCH_CHECK(
      output.is_cuda(),
      "add: expected 'out' to be CUDA tensor, but got tensor on device: ",
      output.device());

  TORCH_CHECK(
      src.is_cuda(),
      "add: expected 'other' to be a CUDA tensor, but got tensor on device: ",
      src.device());

  TORCH_CHECK(
      dense.sizes().equals(src.sizes()),
      "add: expected 'self' and 'other' to have same size, but self has size ",
      dense.sizes(),
      " while other has size ",
      src.sizes(),
      " (FYI: op2-sparse addition does not currently support broadcasting)");

  auto commonDtype = promoteTypes(dense.scalar_type(), src.scalar_type());
  TORCH_CHECK(
      canCast(commonDtype, output.scalar_type()),
      "Can't convert result type ",
      commonDtype,
      " to output ",
      output.scalar_type(),
      " in add operation");

  Tensor src_values = src.values();
  Tensor src_crow_indices = src.crow_indices();
  Tensor src_col_indices = src.col_indices();

  resize_output(output, dense.sizes());

  Tensor resultBuffer = output;
  Tensor valuesBuffer = src_values.to(commonDtype);
  if (output.scalar_type() != commonDtype) {
    resultBuffer = dense.to(commonDtype);
  } else if (!is_same_tensor(output, dense)) {
    resultBuffer.copy_(dense);
  }
  AT_DISPATCH_ALL_TYPES(
      commonDtype,
      "add_out_op2_sparse_csr",
      [&valuesBuffer, &resultBuffer, &alpha, &src_crow_indices, &src_col_indices]() {
        AT_DISPATCH_INDEX_TYPES(
            src_crow_indices.scalar_type(),
            "csr_add_out_crow_indices",
              [&valuesBuffer, &resultBuffer, &alpha, &src_crow_indices, &src_col_indices]() {
                scalar_t* values_accessor = valuesBuffer.data_ptr<scalar_t>();
                scalar_t* out_ptr = resultBuffer.data_ptr<scalar_t>();
                scalar_t cast_value = alpha.to<scalar_t>();

                index_t* crow_indices_accessor = src_crow_indices.data_ptr<index_t>();
                index_t* col_indices_accessor = src_col_indices.data_ptr<index_t>();
                int64_t out_storage_offset = resultBuffer.storage_offset();

                auto out_strides = resultBuffer.strides();
                int64_t out_strides0 = out_strides[0];
                int64_t out_strides1 = out_strides[1];

                hipStream_t stream = at::cuda::getCurrentCUDAStream();
                auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
                auto policy = thrust::cuda::par(allocator).on(stream);

               // Note that this could be wildly imbalanced if the sparsity pattern varies a lot between rows.
               thrust::for_each(
                    policy,
                    thrust::make_counting_iterator(int64_t(0)),
                    thrust::make_counting_iterator(int64_t(src_crow_indices.size(0) - 1)),
                    [values_accessor,
                    crow_indices_accessor,
                    col_indices_accessor,
                    out_ptr,
                    out_storage_offset,
                    out_strides0,
                    cast_value,
                    out_strides1
                    ]__device__(int64_t irow) {
                        int32_t start_index = crow_indices_accessor[irow];
                        int32_t end_index = crow_indices_accessor[irow + 1];

                        for (int i = start_index; i < end_index; ++i) {
                            auto icol = col_indices_accessor[i];
                            auto index = out_storage_offset + irow * out_strides0 + icol * out_strides1;
                            out_ptr[index] += cast_value * values_accessor[i];
                        }
                    });
              });
      });
  if (output.scalar_type() != commonDtype) {
    output.copy_(resultBuffer);
  }
  return output;
}

Tensor& add_out_sparse_csr_cuda(
    const Tensor& self,
    const SparseCsrTensor& other,
    const Scalar& alpha,
    SparseCsrTensor& out) {
  if (self.layout() == kStrided) {
    return add_out_dense_sparse_csr_cuda(out, self, other, alpha);
  } else {
    TORCH_CHECK(
        false,
        "NotImplementedError: Addition of sparse CSR tensors is not yet implemented.")
  }
  return out;
}

} // namespace native
} // namespace at
